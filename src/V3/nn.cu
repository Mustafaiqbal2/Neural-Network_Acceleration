#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define INPUT_SIZE 784
#define HIDDEN_SIZE 128
#define OUTPUT_SIZE 10
#define LEARNING_RATE 0.01f
#define EPOCHS 3
#define BATCH_SIZE 64
#define NUM_CLASSES 10
#define BLOCK_SIZE 256

// Error checking macro
#define CHECK_CUDA_ERROR(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA Error in %s:%d: %s\n", __FILE__, __LINE__, \
                    hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

// Neural network structure for GPU
typedef struct {
    float *d_W1, *d_W2;    // Device weights
    float *d_b1, *d_b2;    // Device biases
    float *h_W1, *h_W2;    // Host weights
    float *h_b1, *h_b2;    // Host biases
} NeuralNetwork;

// Fix 1: Correct batch matrix multiplication kernel
__global__ void batchMatrixMulKernel(float* A, float* B, float* C, float* bias,
                                   int M, int N, int K, int batchSize) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int batch = blockIdx.z;
    
    if (batch < batchSize && row < M && col < K) {
        // For each output element, compute dot product of row of A and column of B
        float sum = bias[row];
        
        // B is now treated as a batch of matrices
        // Each batch element B[batch] is a matrix of shape NxK
        for (int i = 0; i < N; i++) {
            // A[row, i] * B[batch, i, col]
            sum += A[row * N + i] * B[batch * (N * K) + i * K + col];
        }
        
        C[batch * M * K + row * K + col] = sum;
    }
}

// CUDA kernel for ReLU activation for batches
__global__ void batchReluKernel(float* x, int size, int batchSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int batch = blockIdx.y;
    
    if (batch < batchSize && idx < size) {
        x[batch * size + idx] = fmaxf(0.0f, x[batch * size + idx]);
    }
}

// CUDA kernel for applying softmax to each sample in a batch
__global__ void batchSoftmaxKernel(float* x, int size, int batchSize) {
    int batch = blockIdx.x;
    
    if (batch < batchSize) {
        // Get pointer to this batch's data
        float* batch_data = x + batch * size;
        
        // Find maximum (reduce operation)
        float max_val = -INFINITY;
        for (int i = threadIdx.x; i < size; i += blockDim.x) {
            max_val = fmaxf(max_val, batch_data[i]);
        }
        
        // Reduce within block to find maximum
        __shared__ float temp_max[BLOCK_SIZE];
        temp_max[threadIdx.x] = max_val;
        __syncthreads();
        
        for (int stride = blockDim.x/2; stride > 0; stride >>= 1) {
            if (threadIdx.x < stride) {
                temp_max[threadIdx.x] = fmaxf(temp_max[threadIdx.x], temp_max[threadIdx.x + stride]);
            }
            __syncthreads();
        }
        
        float shared_max = temp_max[0];
        
        // Compute exp(x - max) and sum
        float sum = 0.0f;
        for (int i = threadIdx.x; i < size; i += blockDim.x) {
            batch_data[i] = expf(batch_data[i] - shared_max);
            sum += batch_data[i];
        }
        
        // Reduce within block to find sum
        __shared__ float temp_sum[BLOCK_SIZE];
        temp_sum[threadIdx.x] = sum;
        __syncthreads();
        
        for (int stride = blockDim.x/2; stride > 0; stride >>= 1) {
            if (threadIdx.x < stride) {
                temp_sum[threadIdx.x] += temp_sum[threadIdx.x + stride];
            }
            __syncthreads();
        }
        
        float shared_sum = temp_sum[0];
        
        // Normalize by sum
        for (int i = threadIdx.x; i < size; i += blockDim.x) {
            batch_data[i] /= shared_sum;
        }
    }
}

// Add these GPU kernels for calculating loss and accuracy on device
__global__ void calculateBatchLossAccuracy(float* d_batch_output, float* d_batch_target, 
                                         float* d_loss, int* d_correct, int batchSize) {
    __shared__ float batch_loss[BLOCK_SIZE];
    __shared__ int batch_correct[BLOCK_SIZE];
    
    int tid = threadIdx.x;
    int batch_idx = blockIdx.x;
    
    batch_loss[tid] = 0.0f;
    batch_correct[tid] = 0;
    
    if (batch_idx < batchSize) {
        // Each thread handles one sample in the batch
        if (tid == 0) {
            float* output = d_batch_output + batch_idx * OUTPUT_SIZE;
            float* target = d_batch_target + batch_idx * OUTPUT_SIZE;
            
            // Find predicted class
            int pred = 0;
            for (int j = 1; j < OUTPUT_SIZE; j++) {
                if (output[j] > output[pred]) {
                    pred = j;
                }
            }
            
            // Find actual class
            int actual = 0;
            for (int j = 1; j < OUTPUT_SIZE; j++) {
                if (target[j] > target[actual]) {
                    actual = j;
                }
            }
            
            // Compute loss
            for (int j = 0; j < OUTPUT_SIZE; j++) {
                if (target[j] > 0.5f) {
                    batch_loss[tid] -= logf(fmaxf(output[j], 1e-7f));
                }
            }
            
            // Check if prediction was correct
            if (pred == actual) {
                batch_correct[tid] = 1;
            }
        }
    }
    
    __syncthreads();
    
    // Reduce within block
    for (int stride = blockDim.x/2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            batch_loss[tid] += batch_loss[tid + stride];
            batch_correct[tid] += batch_correct[tid + stride];
        }
        __syncthreads();
    }
    
    // Write results back
    if (tid == 0) {
        atomicAdd(d_loss, batch_loss[0]);
        atomicAdd(d_correct, batch_correct[0]);
    }
}

// Allocate memory for a matrix
float** allocateMatrix(int rows, int cols) {
    float** mat = (float**)malloc(rows * sizeof(float*));
    for (int i = 0; i < rows; i++) {
        mat[i] = (float*)malloc(cols * sizeof(float));
    }
    return mat;
}

// Initialize neural network
NeuralNetwork* createNetwork() {
    NeuralNetwork* net = (NeuralNetwork*)malloc(sizeof(NeuralNetwork));
    
    // Allocate host memory
    net->h_W1 = (float*)malloc(HIDDEN_SIZE * INPUT_SIZE * sizeof(float));
    net->h_W2 = (float*)malloc(OUTPUT_SIZE * HIDDEN_SIZE * sizeof(float));
    net->h_b1 = (float*)malloc(HIDDEN_SIZE * sizeof(float));
    net->h_b2 = (float*)malloc(OUTPUT_SIZE * sizeof(float));

    // Update in createNetwork function
    // Initialize weights with Xavier/Glorot initialization
    float w1_scale = sqrtf(6.0f / (INPUT_SIZE + HIDDEN_SIZE));
    float w2_scale = sqrtf(6.0f / (HIDDEN_SIZE + OUTPUT_SIZE));

    for (int i = 0; i < HIDDEN_SIZE * INPUT_SIZE; i++)
        net->h_W1[i] = ((2.0f * (float)rand() / RAND_MAX) - 1.0f) * w1_scale;
    for (int i = 0; i < OUTPUT_SIZE * HIDDEN_SIZE; i++)
        net->h_W2[i] = ((2.0f * (float)rand() / RAND_MAX) - 1.0f) * w2_scale;
    for (int i = 0; i < HIDDEN_SIZE; i++)
        net->h_b1[i] = 0.0f;
    for (int i = 0; i < OUTPUT_SIZE; i++)
        net->h_b2[i] = 0.0f;

    // Allocate device memory
    CHECK_CUDA_ERROR(hipMalloc(&net->d_W1, HIDDEN_SIZE * INPUT_SIZE * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&net->d_W2, OUTPUT_SIZE * HIDDEN_SIZE * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&net->d_b1, HIDDEN_SIZE * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&net->d_b2, OUTPUT_SIZE * sizeof(float)));

    // Copy data to device
    CHECK_CUDA_ERROR(hipMemcpy(net->d_W1, net->h_W1, 
        HIDDEN_SIZE * INPUT_SIZE * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(net->d_W2, net->h_W2,
        OUTPUT_SIZE * HIDDEN_SIZE * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(net->d_b1, net->h_b1,
        HIDDEN_SIZE * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(net->d_b2, net->h_b2,
        OUTPUT_SIZE * sizeof(float), hipMemcpyHostToDevice));

    return net;
}

// Batch version of computeGradients kernel
__global__ void batchComputeGradients(float* d_batch_output, float* d_batch_target, 
                                    float* d_batch_hidden, float* d_batch_input,
                                    float* d_W2_grad, float* d_b2_grad,
                                    float* d_W1_grad, float* d_b1_grad,
                                    float* d_W2, float* d_W1,
                                    int batchSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int batch = blockIdx.y;
    
    if (batch >= batchSize) return;
    
    // Get pointers to this batch's data
    float* output = d_batch_output + batch * OUTPUT_SIZE;
    float* target = d_batch_target + batch * OUTPUT_SIZE;
    float* hidden = d_batch_hidden + batch * HIDDEN_SIZE;
    float* input = d_batch_input + batch * INPUT_SIZE;
    
    // Output layer gradients
    if (idx < OUTPUT_SIZE) {
        float output_error = output[idx] - target[idx];
        
        // Accumulate bias gradients (one per batch item)
        atomicAdd(&d_b2_grad[idx], output_error);
        
        // Accumulate weight gradients
        for (int j = 0; j < HIDDEN_SIZE; j++) {
            atomicAdd(&d_W2_grad[idx * HIDDEN_SIZE + j], output_error * hidden[j]);
        }
    }
    
    // Hidden layer gradients
    if (idx < HIDDEN_SIZE) {
        float hidden_error = 0.0f;
        
        // Calculate error at hidden layer
        for (int j = 0; j < OUTPUT_SIZE; j++) {
            hidden_error += (output[j] - target[j]) * d_W2[j * HIDDEN_SIZE + idx];
        }
        
        // Apply ReLU derivative
        hidden_error *= (hidden[idx] > 0.0f);
        
        // Accumulate bias gradients
        atomicAdd(&d_b1_grad[idx], hidden_error);
        
        // Accumulate weight gradients
        for (int j = 0; j < INPUT_SIZE; j++) {
            atomicAdd(&d_W1_grad[idx * INPUT_SIZE + j], hidden_error * input[j]);
        }
    }
}

// Batch version of parameter updates
__global__ void batchUpdateParameters(float* d_W, float* d_b, float* d_W_grad, 
                                    float* d_b_grad, int rows, int cols, 
                                    float learning_rate, int batchSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Update weights
    if (idx < rows * cols) {
        // Normalize by batch size and apply learning rate
        d_W[idx] -= (learning_rate / batchSize) * d_W_grad[idx];
        // Reset gradient for next batch
        d_W_grad[idx] = 0.0f;
    }
    
    // Update biases
    if (idx < rows) {
        // Normalize by batch size and apply learning rate
        d_b[idx] -= (learning_rate / batchSize) * d_b_grad[idx];
        // Reset gradient for next batch
        d_b_grad[idx] = 0.0f;
    }
}

// New batched backward function
void batchBackward(NeuralNetwork* net, float* d_batch_input, float* d_batch_hidden, 
                  float* d_batch_output, float* d_batch_target, int batchSize) {
    // Allocate gradient matrices
    static float *d_W1_grad, *d_W2_grad, *d_b1_grad, *d_b2_grad;
    static bool gradients_initialized = false;
    
    // Initialize gradient buffers if first call
    if (!gradients_initialized) {
        CHECK_CUDA_ERROR(hipMalloc(&d_W1_grad, HIDDEN_SIZE * INPUT_SIZE * sizeof(float)));
        CHECK_CUDA_ERROR(hipMalloc(&d_W2_grad, OUTPUT_SIZE * HIDDEN_SIZE * sizeof(float)));
        CHECK_CUDA_ERROR(hipMalloc(&d_b1_grad, HIDDEN_SIZE * sizeof(float)));
        CHECK_CUDA_ERROR(hipMalloc(&d_b2_grad, OUTPUT_SIZE * sizeof(float)));
        
        gradients_initialized = true;
    }
    
    // Clear gradients before accumulating new ones - this is critical!
    hipMemset(d_W1_grad, 0, HIDDEN_SIZE * INPUT_SIZE * sizeof(float));
    hipMemset(d_W2_grad, 0, OUTPUT_SIZE * HIDDEN_SIZE * sizeof(float));
    hipMemset(d_b1_grad, 0, HIDDEN_SIZE * sizeof(float));
    hipMemset(d_b2_grad, 0, OUTPUT_SIZE * sizeof(float));
    
    // Compute gradients for batch
    dim3 blockDim(256);
    dim3 gridDim(max(1, (max(OUTPUT_SIZE, HIDDEN_SIZE) + blockDim.x - 1) / blockDim.x), batchSize);
    
    batchComputeGradients<<<gridDim, blockDim>>>(
        d_batch_output, d_batch_target, d_batch_hidden, d_batch_input,
        d_W2_grad, d_b2_grad, d_W1_grad, d_b1_grad,
        net->d_W2, net->d_W1, batchSize
    );
    
    // Update parameters
    dim3 updateBlockDim(256);
    dim3 updateGridDim((max(HIDDEN_SIZE * INPUT_SIZE, OUTPUT_SIZE * HIDDEN_SIZE) + 
                      updateBlockDim.x - 1) / updateBlockDim.x);
                      
    batchUpdateParameters<<<updateGridDim, updateBlockDim>>>(
        net->d_W1, net->d_b1, d_W1_grad, d_b1_grad,
        HIDDEN_SIZE, INPUT_SIZE, LEARNING_RATE, batchSize
    );
    
    batchUpdateParameters<<<updateGridDim, updateBlockDim>>>(
        net->d_W2, net->d_b2, d_W2_grad, d_b2_grad,
        OUTPUT_SIZE, HIDDEN_SIZE, LEARNING_RATE, batchSize
    );
}

// Remove debug messages and error checks that aren't needed anymore
void forwardBatch(NeuralNetwork* net, float* d_batch_input, float* d_batch_hidden, 
                 float* d_batch_output, int batchSize, hipStream_t stream = 0) {
    // First layer: Hidden = W1 * Input + b1
    dim3 blockDim(16, 16);
    
    // For hidden layer, output is of size (batchSize, HIDDEN_SIZE)
    dim3 gridDim;
    
    // K=1 for matrix-vector multiplication (1 column per result)
    gridDim.x = 1;  // K=1 so only 1 column
    gridDim.y = (HIDDEN_SIZE + blockDim.y - 1) / blockDim.y;  // Rows
    gridDim.z = batchSize;  // Batch dimension
    
    // First layer: FC + ReLU
    batchMatrixMulKernel<<<gridDim, blockDim, 0, stream>>>(
        net->d_W1, d_batch_input, d_batch_hidden, 
        net->d_b1, HIDDEN_SIZE, INPUT_SIZE, 1, batchSize);
    
    // Apply ReLU to the hidden layer outputs
    dim3 reluGrid((HIDDEN_SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE, batchSize);
    batchReluKernel<<<reluGrid, BLOCK_SIZE, 0, stream>>>(
        d_batch_hidden, HIDDEN_SIZE, batchSize);
    
    // Output layer: Output = W2 * Hidden + b2
    gridDim.x = 1;  // K=1 (single column output)
    gridDim.y = (OUTPUT_SIZE + blockDim.y - 1) / blockDim.y;  // Rows
    gridDim.z = batchSize;  // Batch dimension
    
    batchMatrixMulKernel<<<gridDim, blockDim, 0, stream>>>(
        net->d_W2, d_batch_hidden, d_batch_output,
        net->d_b2, OUTPUT_SIZE, HIDDEN_SIZE, 1, batchSize);
    
    // Apply softmax to each sample in the batch
    batchSoftmaxKernel<<<batchSize, BLOCK_SIZE, 0, stream>>>(
        d_batch_output, OUTPUT_SIZE, batchSize);
}

// Modified train function with full batch processing
void train(NeuralNetwork* net, float** h_images, float** h_labels, int numImages) {
    const int batchSize = BATCH_SIZE;
    const int numBatches = (numImages + batchSize - 1) / batchSize;
    
    // Allocate device memory for batches
    float *d_batch_input, *d_batch_hidden, *d_batch_output, *d_batch_target;
    CHECK_CUDA_ERROR(hipMalloc(&d_batch_input, batchSize * INPUT_SIZE * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_batch_hidden, batchSize * HIDDEN_SIZE * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_batch_output, batchSize * OUTPUT_SIZE * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_batch_target, batchSize * OUTPUT_SIZE * sizeof(float)));
    
    // Allocate device memory for metrics
    float *d_loss;
    int *d_correct;
    CHECK_CUDA_ERROR(hipMalloc(&d_loss, sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_correct, sizeof(int)));
    
    // Use page-locked memory for faster transfers
    float *h_batch_data;  // Combined buffer for both input and target data
    size_t input_bytes = batchSize * INPUT_SIZE * sizeof(float);
    size_t target_bytes = batchSize * OUTPUT_SIZE * sizeof(float);
    CHECK_CUDA_ERROR(hipHostMalloc(&h_batch_data, input_bytes + target_bytes));
    
    // Create pointers to the appropriate sections of the combined buffer
    float *h_batch_input = h_batch_data;
    float *h_batch_target = h_batch_data + batchSize * INPUT_SIZE;
    
    // Create and initialize index array for shuffling
    int* indices = (int*)malloc(numImages * sizeof(int));
    for (int i = 0; i < numImages; i++) {
        indices[i] = i;
    }
    
    // Set up CUDA streams for overlapping operations
    hipStream_t stream;
    hipStreamCreate(&stream);
    
    for (int epoch = 0; epoch < EPOCHS; epoch++) {
        // Reset metrics
        float h_loss = 0.0f;
        int h_correct = 0;
        CHECK_CUDA_ERROR(hipMemcpy(d_loss, &h_loss, sizeof(float), hipMemcpyHostToDevice));
        CHECK_CUDA_ERROR(hipMemcpy(d_correct, &h_correct, sizeof(int), hipMemcpyHostToDevice));
        
        clock_t epoch_start = clock();
        
        // Shuffle data indices for this epoch
        for (int i = numImages - 1; i > 0; i--) {
            int j = rand() % (i + 1);
            int temp = indices[i];
            indices[i] = indices[j];
            indices[j] = temp;
        }
        
        // Train in batches
        for (int batch = 0; batch < numBatches; batch++) {
            int start_idx = batch * batchSize;
            int current_batch_size = min(batchSize, numImages - start_idx);
            
            // Pack all batch data at once (input and target together)
            for (int i = 0; i < current_batch_size; i++) {
                int idx = indices[start_idx + i];
                
                // Copy input data
                memcpy(h_batch_input + i * INPUT_SIZE, 
                       h_images[idx], 
                       INPUT_SIZE * sizeof(float));
                       
                // Copy target data
                memcpy(h_batch_target + i * OUTPUT_SIZE, 
                       h_labels[idx], 
                       OUTPUT_SIZE * sizeof(float));
            }
            
            // Combine transfers using a single stream for pipelining
            hipMemcpyAsync(d_batch_input, h_batch_input,
                current_batch_size * INPUT_SIZE * sizeof(float), 
                hipMemcpyHostToDevice, stream);
                
            hipMemcpyAsync(d_batch_target, h_batch_target,
                current_batch_size * OUTPUT_SIZE * sizeof(float), 
                hipMemcpyHostToDevice, stream);
                
            // Ensure transfers complete before proceeding
            hipStreamSynchronize(stream);
                
            // Forward pass
            forwardBatch(net, d_batch_input, d_batch_hidden, d_batch_output, current_batch_size, stream);
            
            // Calculate metrics on GPU
            calculateBatchLossAccuracy<<<current_batch_size, BLOCK_SIZE, 0, stream>>>(
                d_batch_output, d_batch_target, d_loss, d_correct, current_batch_size);
            
            // Backward pass
            batchBackward(net, d_batch_input, d_batch_hidden, d_batch_output, d_batch_target, current_batch_size);
        }
        
        // Get final metrics
        CHECK_CUDA_ERROR(hipMemcpy(&h_loss, d_loss, sizeof(float), hipMemcpyDeviceToHost));
        CHECK_CUDA_ERROR(hipMemcpy(&h_correct, d_correct, sizeof(int), hipMemcpyDeviceToHost));
        
        printf("Epoch %d - Loss: %.4f - Train Accuracy: %.2f%% - Time: %.3fs\n",
               epoch + 1, h_loss / numImages, (h_correct / (float)numImages) * 100,
               (float)(clock() - epoch_start) / CLOCKS_PER_SEC);
    }
    
    // Clean up
    hipStreamDestroy(stream);
    hipFree(d_batch_input);
    hipFree(d_batch_hidden);
    hipFree(d_batch_output);
    hipFree(d_batch_target);
    hipFree(d_loss);
    hipFree(d_correct);
    hipHostFree(h_batch_data);  // Free the combined buffer
    free(indices);
}

// Free network memory
void freeNetwork(NeuralNetwork* net) {
    // Free host memory
    free(net->h_W1);
    free(net->h_W2);
    free(net->h_b1);
    free(net->h_b2);

    // Free device memory
    hipFree(net->d_W1);
    hipFree(net->d_W2);
    hipFree(net->d_b1);
    hipFree(net->d_b2);

    free(net);
}

// Updated evaluate function
void evaluate(NeuralNetwork* net, float** h_images, float** h_labels, int numImages) {
    const int batchSize = BATCH_SIZE;
    
    // Allocate device memory for batches
    float *d_batch_input, *d_batch_hidden, *d_batch_output, *d_batch_target;
    CHECK_CUDA_ERROR(hipMalloc(&d_batch_input, batchSize * INPUT_SIZE * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_batch_hidden, batchSize * HIDDEN_SIZE * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_batch_output, batchSize * OUTPUT_SIZE * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_batch_target, batchSize * OUTPUT_SIZE * sizeof(float)));
    
    // For loss and accuracy tracking
    float *d_loss;
    int *d_correct;
    CHECK_CUDA_ERROR(hipMalloc(&d_loss, sizeof(float)));  // Allocate dummy loss buffer
    CHECK_CUDA_ERROR(hipMalloc(&d_correct, sizeof(int)));
    
    // Initialize counters
    float h_loss = 0.0f;
    int h_correct = 0;
    CHECK_CUDA_ERROR(hipMemcpy(d_loss, &h_loss, sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_correct, &h_correct, sizeof(int), hipMemcpyHostToDevice));
    
    // Use pinned memory for faster transfers
    float *h_batch_input, *h_batch_target;
    CHECK_CUDA_ERROR(hipHostMalloc(&h_batch_input, batchSize * INPUT_SIZE * sizeof(float)));
    CHECK_CUDA_ERROR(hipHostMalloc(&h_batch_target, batchSize * OUTPUT_SIZE * sizeof(float)));
    
    // Process test data in batches
    for (int batch_start = 0; batch_start < numImages; batch_start += batchSize) {
        int current_batch_size = min(batchSize, numImages - batch_start);
        
        // Pack batch data 
        for (int i = 0; i < current_batch_size; i++) {
            memcpy(h_batch_input + i * INPUT_SIZE, 
                   h_images[batch_start + i],
                   INPUT_SIZE * sizeof(float));
            
            memcpy(h_batch_target + i * OUTPUT_SIZE,
                   h_labels[batch_start + i],
                   OUTPUT_SIZE * sizeof(float));
        }
        
        // Copy to device
        CHECK_CUDA_ERROR(hipMemcpy(d_batch_input, h_batch_input,
            current_batch_size * INPUT_SIZE * sizeof(float), hipMemcpyHostToDevice));
        CHECK_CUDA_ERROR(hipMemcpy(d_batch_target, h_batch_target,
            current_batch_size * OUTPUT_SIZE * sizeof(float), hipMemcpyHostToDevice));
        
        // Forward pass
        forwardBatch(net, d_batch_input, d_batch_hidden, d_batch_output, current_batch_size);
        
        // Use d_loss instead of nullptr - this was causing the error
        calculateBatchLossAccuracy<<<current_batch_size, BLOCK_SIZE>>>(
            d_batch_output, d_batch_target, d_loss, d_correct, current_batch_size);
    }
    
    // Get final accuracy
    CHECK_CUDA_ERROR(hipMemcpy(&h_correct, d_correct, sizeof(int), hipMemcpyDeviceToHost));
    
    // Clean up resources
    hipFree(d_batch_input);
    hipFree(d_batch_hidden);
    hipFree(d_batch_output);
    hipFree(d_batch_target);
    hipFree(d_loss);
    hipFree(d_correct);
    hipHostFree(h_batch_input);
    hipHostFree(h_batch_target);
    
    printf("Test Accuracy: %.2f%%\n", (h_correct / (float)numImages) * 100);
}

// Read MNIST dataset
float** loadMNISTImages(const char* filename, int numImages) {
    FILE* file = fopen(filename, "rb");
    if (!file) {
        printf("Error opening %s\n", filename);
        exit(1);
    }
    fseek(file, 16, SEEK_SET);
    float** images = allocateMatrix(numImages, INPUT_SIZE);
    for (int i = 0; i < numImages; i++) {
        for (int j = 0; j < INPUT_SIZE; j++) {
            unsigned char pixel;

            // fread(&pixel, sizeof(unsigned char), 1, file);
            if (fread(&pixel, sizeof(unsigned char), 1, file) != 1) {
                fprintf(stderr, "Error: Failed to read pixel\n");
                fclose(file);
                exit(EXIT_FAILURE);
            }

            images[i][j] = pixel / 255.0;
        }
    }
    fclose(file);
    return images;
}


float** loadMNISTLabels(const char* filename, int numLabels) {
    FILE* file = fopen(filename, "rb");
    if (!file) {
        printf("Error opening %s\n", filename);
        exit(1);
    }
    fseek(file, 8, SEEK_SET);
    float** labels = allocateMatrix(numLabels, OUTPUT_SIZE);
    for (int i = 0; i < numLabels; i++) {
        unsigned char label;
        // fread(&label, sizeof(unsigned char), 1, file);
        if (fread(&label, sizeof(unsigned char), 1, file) != 1) {
            fprintf(stderr, "Error: Failed to read label\n");
            fclose(file);
            exit(EXIT_FAILURE);
        }

        for (int j = 0; j < OUTPUT_SIZE; j++) {
            labels[i][j] = (j == label) ? 1.0 : 0.0;
        }
    }
    fclose(file);
    return labels;
}


// Main function remains similar but with float arrays instead of double
int main() {
    printf("MNIST Neural Network\n\n");
    // Start measuring total execution time
    clock_t total_start = clock();

    // Measure time for loading data
    clock_t start = clock();
    float** train_images = loadMNISTImages("../data/train-images.idx3-ubyte", 60000);
    float** train_labels = loadMNISTLabels("../data/train-labels.idx1-ubyte", 60000);
    float** test_images = loadMNISTImages("../data/t10k-images.idx3-ubyte", 10000);
    float** test_labels = loadMNISTLabels("../data/t10k-labels.idx1-ubyte", 10000);
    clock_t end = clock();
    printf("Time to load data: %.3fs\n", (double)(end - start) / CLOCKS_PER_SEC);

    // Measure time for training
    start = clock();
    NeuralNetwork* net = createNetwork();
    train(net, train_images, train_labels, 60000);
    end = clock();
    printf("Time to train: %.3fs\n", (double)(end - start) / CLOCKS_PER_SEC);

    // Measure time for evaluation
    start = clock();
    evaluate(net, test_images, test_labels, 10000);
    end = clock();
    printf("Time to evaluate: %.3fs\n", (double)(end - start) / CLOCKS_PER_SEC);

    // End measuring total execution time
    clock_t total_end = clock();
    printf("Total execution time: %.3fs\n", (double)(total_end - total_start) / CLOCKS_PER_SEC);

    freeNetwork(net);
    return 0;
}
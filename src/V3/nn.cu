#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>

#define INPUT_SIZE 784
#define HIDDEN_SIZE 128
#define OUTPUT_SIZE 10
#define LEARNING_RATE 0.01f
#define EPOCHS 3
#define BATCH_SIZE 4096
#define NUM_CLASSES 10
#define BLOCK_SIZE 256

// Error checking macro
#define CHECK_CUDA_ERROR(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA Error in %s:%d: %s\n", __FILE__, __LINE__, \
                    hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

// Neural network structure for GPU
typedef struct {
    float *d_W1, *d_W2;    // Device weights
    float *d_b1, *d_b2;    // Device biases
    float *h_W1, *h_W2;    // Host weights
    float *h_b1, *h_b2;    // Host biases
} NeuralNetwork;

// Fused kernel for matrix multiplication + ReLU activation
__global__ void batchFCReluKernel(float* weights, float* inputs, float* outputs, float* bias,
                               int output_size, int input_size, int batch_size) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int batch = blockIdx.z;
    
    if (batch < batch_size && row < output_size) {
        float sum = bias[row];
        
        for (int i = 0; i < input_size; i++) {
            sum += weights[row * input_size + i] * inputs[batch * input_size + i];
        }
        
        // Apply ReLU directly
        outputs[batch * output_size + row] = fmaxf(0.0f, sum);
    }
}

// Kernel for matrix multiplication + linear (no activation)
__global__ void batchFCKernel(float* weights, float* inputs, float* outputs, float* bias,
                           int output_size, int input_size, int batch_size) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int batch = blockIdx.z;
    
    if (batch < batch_size && row < output_size) {
        float sum = bias[row];
        
        for (int i = 0; i < input_size; i++) {
            sum += weights[row * input_size + i] * inputs[batch * input_size + i];
        }
        
        // No activation
        outputs[batch * output_size + row] = sum;
    }
}



// Optimized softmax kernel with register caching
__global__ void batchSoftmaxSmallKernel(float* x, int size, int batchSize) {
    int batch = blockIdx.x;
    int tid = threadIdx.x;
    
    if (batch < batchSize) {
        // Cache batch offset in register
        float* batch_data = x + batch * size;
        
        // Use shared memory for this small array
        __shared__ float data[32];
        __shared__ float max_val;
        __shared__ float sum_val;
        
        // Load data into shared memory (and cache our value in register)
        float my_val = 0.0f;
        if (tid < size) {
            my_val = batch_data[tid];
            data[tid] = my_val;
        }
        __syncthreads();
        
        // Find maximum with thread 0
        if (tid == 0) {
            max_val = data[0];
            for (int i = 1; i < size; i++) {
                max_val = fmaxf(max_val, data[i]);
            }
        }
        __syncthreads();
        
        // Cache max value in register
        float max_val_reg = max_val;
        
        // Compute exp(x - max) and prepare for sum
        if (tid < size) {
            float exp_val = expf(my_val - max_val_reg);
            data[tid] = exp_val;
        }
        __syncthreads();
        
        // Compute sum with reduction
        if (tid == 0) {
            sum_val = 0.0f;
            for (int i = 0; i < size; i++) {
                sum_val += data[i];
            }
        }
        __syncthreads();
        
        // Cache sum in register
        float sum_val_reg = sum_val;
        
        // Normalize and write back
        if (tid < size) {
            batch_data[tid] = data[tid] / sum_val_reg;
        }
    }
}

// Add these GPU kernels for calculating loss and accuracy on device
__global__ void calculateBatchLossAccuracy(float* d_batch_output, float* d_batch_target, 
                                         float* d_loss, int* d_correct, int batchSize) {
    __shared__ float batch_loss[BLOCK_SIZE];
    __shared__ int batch_correct[BLOCK_SIZE];
    
    int tid = threadIdx.x;
    int batch_idx = blockIdx.x;
    
    batch_loss[tid] = 0.0f;
    batch_correct[tid] = 0;
    
    if (batch_idx < batchSize) {
        // Each thread handles one sample in the batch
        if (tid == 0) {
            float* output = d_batch_output + batch_idx * OUTPUT_SIZE;
            float* target = d_batch_target + batch_idx * OUTPUT_SIZE;
            
            // Find predicted class
            int pred = 0;
            for (int j = 1; j < OUTPUT_SIZE; j++) {
                if (output[j] > output[pred]) {
                    pred = j;
                }
            }
            
            // Find actual class
            int actual = 0;
            for (int j = 1; j < OUTPUT_SIZE; j++) {
                if (target[j] > target[actual]) {
                    actual = j;
                }
            }
            
            // Compute loss
            for (int j = 0; j < OUTPUT_SIZE; j++) {
                if (target[j] > 0.5f) {
                    batch_loss[tid] -= logf(fmaxf(output[j], 1e-7f));
                }
            }
            
            // Check if prediction was correct
            if (pred == actual) {
                batch_correct[tid] = 1;
            }
        }
    }
    
    __syncthreads();
    
    // Reduce within block
    for (int stride = blockDim.x/2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            batch_loss[tid] += batch_loss[tid + stride];
            batch_correct[tid] += batch_correct[tid + stride];
        }
        __syncthreads();
    }
    
    // Write results back
    if (tid == 0) {
        atomicAdd(d_loss, batch_loss[0]);
        atomicAdd(d_correct, batch_correct[0]);
    }
}

// Allocate memory for a matrix
float** allocateMatrix(int rows, int cols) {
    float** mat = (float**)malloc(rows * sizeof(float*));
    for (int i = 0; i < rows; i++) {
        mat[i] = (float*)malloc(cols * sizeof(float));
    }
    return mat;
}

// Enhanced createNetwork with CPU-optimized initialization
NeuralNetwork* createNetwork() {
    NeuralNetwork* net = (NeuralNetwork*)malloc(sizeof(NeuralNetwork));
    
    // Allocate host memory
    net->h_W1 = (float*)aligned_alloc(32, HIDDEN_SIZE * INPUT_SIZE * sizeof(float));
    net->h_W2 = (float*)aligned_alloc(32, OUTPUT_SIZE * HIDDEN_SIZE * sizeof(float));
    net->h_b1 = (float*)aligned_alloc(32, HIDDEN_SIZE * sizeof(float));
    net->h_b2 = (float*)aligned_alloc(32, OUTPUT_SIZE * sizeof(float));

    // Initialize weights with Xavier/Glorot on CPU (more efficient than GPU for this)
    float w1_scale = sqrtf(6.0f / (INPUT_SIZE + HIDDEN_SIZE));
    float w2_scale = sqrtf(6.0f / (HIDDEN_SIZE + OUTPUT_SIZE));

    // Use OpenMP for parallel initialization
    #pragma omp parallel for
    for (int i = 0; i < HIDDEN_SIZE * INPUT_SIZE; i++)
        net->h_W1[i] = ((2.0f * (float)rand() / RAND_MAX) - 1.0f) * w1_scale;
        
    #pragma omp parallel for
    for (int i = 0; i < OUTPUT_SIZE * HIDDEN_SIZE; i++)
        net->h_W2[i] = ((2.0f * (float)rand() / RAND_MAX) - 1.0f) * w2_scale;
        
    #pragma omp parallel for
    for (int i = 0; i < HIDDEN_SIZE; i++)
        net->h_b1[i] = 0.0f;
        
    #pragma omp parallel for
    for (int i = 0; i < OUTPUT_SIZE; i++)
        net->h_b2[i] = 0.0f;

    // Allocate device memory and copy from host
    CHECK_CUDA_ERROR(hipMalloc(&net->d_W1, HIDDEN_SIZE * INPUT_SIZE * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&net->d_W2, OUTPUT_SIZE * HIDDEN_SIZE * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&net->d_b1, HIDDEN_SIZE * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&net->d_b2, OUTPUT_SIZE * sizeof(float)));

    // Copy data to device
    CHECK_CUDA_ERROR(hipMemcpy(net->d_W1, net->h_W1, 
        HIDDEN_SIZE * INPUT_SIZE * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(net->d_W2, net->h_W2,
        OUTPUT_SIZE * HIDDEN_SIZE * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(net->d_b1, net->h_b1,
        HIDDEN_SIZE * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(net->d_b2, net->h_b2,
        OUTPUT_SIZE * sizeof(float), hipMemcpyHostToDevice));

    return net;
}

// Optimized gradient computation kernel
__global__ void batchComputeGradientsOptimized(float* d_batch_output, float* d_batch_target, 
                                           float* d_batch_hidden, float* d_batch_input,
                                           float* d_W2_grad, float* d_b2_grad,
                                           float* d_W1_grad, float* d_b1_grad,
                                           float* d_W2, float* d_W1,
                                           int batchSize) {
    int tid = threadIdx.x;
    int batch = blockIdx.x;
    
    if (batch >= batchSize) return;
    
    // Get pointers to this batch's data
    float* output = d_batch_output + batch * OUTPUT_SIZE;
    float* target = d_batch_target + batch * OUTPUT_SIZE;
    float* hidden = d_batch_hidden + batch * HIDDEN_SIZE;
    float* input = d_batch_input + batch * INPUT_SIZE;
    
    // Use shared memory to compute and store output errors
    __shared__ float output_errors[32];  // Enough for OUTPUT_SIZE=10
    
    // Calculate output errors in parallel
    if (tid < OUTPUT_SIZE) {
        output_errors[tid] = output[tid] - target[tid];
        
        // Update bias gradient
        atomicAdd(&d_b2_grad[tid], output_errors[tid]);
    }
    __syncthreads();
    
    // Each thread updates a subset of W2 gradients
    for (int i = tid; i < OUTPUT_SIZE * HIDDEN_SIZE; i += blockDim.x) {
        int out_idx = i / HIDDEN_SIZE;
        int hid_idx = i % HIDDEN_SIZE;
        atomicAdd(&d_W2_grad[i], output_errors[out_idx] * hidden[hid_idx]);
    }
    
    // Compute hidden layer errors
    __shared__ float hidden_errors[128];  // Maximum HIDDEN_SIZE we'd expect
    
    // First initialize all to zero
    for (int i = tid; i < HIDDEN_SIZE; i += blockDim.x) {
        hidden_errors[i] = 0.0f;
    }
    __syncthreads();
    
    // Compute hidden errors in parallel across threads
    for (int i = 0; i < OUTPUT_SIZE; i++) {
        for (int j = tid; j < HIDDEN_SIZE; j += blockDim.x) {
            hidden_errors[j] += output_errors[i] * d_W2[i * HIDDEN_SIZE + j];
        }
    }
    __syncthreads();
    
    // Apply ReLU derivative and update bias gradients
    for (int i = tid; i < HIDDEN_SIZE; i += blockDim.x) {
        hidden_errors[i] *= (hidden[i] > 0.0f);
        atomicAdd(&d_b1_grad[i], hidden_errors[i]);
    }
    __syncthreads();
    
    // Each thread updates a subset of W1 gradients
    for (int i = tid; i < HIDDEN_SIZE * INPUT_SIZE; i += blockDim.x) {
        int hid_idx = i / INPUT_SIZE;
        int in_idx = i % INPUT_SIZE;
        atomicAdd(&d_W1_grad[i], hidden_errors[hid_idx] * input[in_idx]);
    }
}

// Optimized parameter update kernel
__global__ void batchUpdateParametersOptimized(float* d_W, float* d_b, float* d_W_grad, 
                                           float* d_b_grad, int rows, int cols, 
                                           float learning_rate, int batchSize) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Each thread updates multiple weights using grid-stride loop
    for (int i = tid; i < rows * cols; i += blockDim.x * gridDim.x) {
        d_W[i] -= (learning_rate / batchSize) * d_W_grad[i];
        d_W_grad[i] = 0.0f;  // Reset gradient for next batch
    }
    
    // Update biases with efficient access pattern
    if (tid < rows) {
        d_b[tid] -= (learning_rate / batchSize) * d_b_grad[tid];
        d_b_grad[tid] = 0.0f;  // Reset gradient for next batch
    }
}

// Optimized backward pass using more efficient kernels
void batchBackward(NeuralNetwork* net, float* d_batch_input, float* d_batch_hidden, 
                  float* d_batch_output, float* d_batch_target, int batchSize) {
    // Allocate gradient matrices
    static float *d_W1_grad, *d_W2_grad, *d_b1_grad, *d_b2_grad;
    static bool gradients_initialized = false;
    
    // Initialize gradient buffers if first call
    if (!gradients_initialized) {
        CHECK_CUDA_ERROR(hipMalloc(&d_W1_grad, HIDDEN_SIZE * INPUT_SIZE * sizeof(float)));
        CHECK_CUDA_ERROR(hipMalloc(&d_W2_grad, OUTPUT_SIZE * HIDDEN_SIZE * sizeof(float)));
        CHECK_CUDA_ERROR(hipMalloc(&d_b1_grad, HIDDEN_SIZE * sizeof(float)));
        CHECK_CUDA_ERROR(hipMalloc(&d_b2_grad, OUTPUT_SIZE * sizeof(float)));
        
        gradients_initialized = true;
    }
    
    // Clear gradients
    hipMemset(d_W1_grad, 0, HIDDEN_SIZE * INPUT_SIZE * sizeof(float));
    hipMemset(d_W2_grad, 0, OUTPUT_SIZE * HIDDEN_SIZE * sizeof(float));
    hipMemset(d_b1_grad, 0, HIDDEN_SIZE * sizeof(float));
    hipMemset(d_b2_grad, 0, OUTPUT_SIZE * sizeof(float));
    
    // Compute gradients with optimized kernel - one block per batch item
    dim3 blockDim(256);
    dim3 gridDim(batchSize);
    
    batchComputeGradientsOptimized<<<gridDim, blockDim>>>(
        d_batch_output, d_batch_target, d_batch_hidden, d_batch_input,
        d_W2_grad, d_b2_grad, d_W1_grad, d_b1_grad,
        net->d_W2, net->d_W1, batchSize
    );
    
    // Update parameters with optimized kernel
    batchUpdateParametersOptimized<<<32, 256>>>(
        net->d_W1, net->d_b1, d_W1_grad, d_b1_grad,
        HIDDEN_SIZE, INPUT_SIZE, LEARNING_RATE, batchSize
    );
    
    batchUpdateParametersOptimized<<<32, 256>>>(
        net->d_W2, net->d_b2, d_W2_grad, d_b2_grad,
        OUTPUT_SIZE, HIDDEN_SIZE, LEARNING_RATE, batchSize
    );
}

// Optimized forward pass using kernel fusion
void forwardBatch(NeuralNetwork* net, float* d_batch_input, float* d_batch_hidden, 
                 float* d_batch_output, int batchSize, hipStream_t stream = 0) {
    // Set block dimensions for FC layers
    dim3 blockDim(1, 256);  // Using y dimension for row parallelism
    
    // First layer: FC + ReLU fused
    dim3 gridDim1(1, (HIDDEN_SIZE + blockDim.y - 1) / blockDim.y, batchSize);
    batchFCReluKernel<<<gridDim1, blockDim, 0, stream>>>(
        net->d_W1, d_batch_input, d_batch_hidden, 
        net->d_b1, HIDDEN_SIZE, INPUT_SIZE, batchSize);
    
    // Second layer: FC (no activation, will apply softmax after)
    dim3 gridDim2(1, (OUTPUT_SIZE + blockDim.y - 1) / blockDim.y, batchSize);
    batchFCKernel<<<gridDim2, blockDim, 0, stream>>>(
        net->d_W2, d_batch_hidden, d_batch_output,
        net->d_b2, OUTPUT_SIZE, HIDDEN_SIZE, batchSize);
    
    // Apply optimized softmax for small output vectors
    batchSoftmaxSmallKernel<<<batchSize, 32, 0, stream>>>(
        d_batch_output, OUTPUT_SIZE, batchSize);
}

// Modified train function with prefetching
void train(NeuralNetwork* net, float** h_images, float** h_labels, int numImages) {
    const int batchSize = BATCH_SIZE;
    const int numBatches = (numImages + batchSize - 1) / batchSize;
    
    // CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    
    // Double buffering for prefetching
    float *d_batch_input[2], *d_batch_hidden[2], *d_batch_output[2], *d_batch_target[2];
    
    // Allocate two sets of device memory for double buffering
    for (int i = 0; i < 2; i++) {
        CHECK_CUDA_ERROR(hipMalloc(&d_batch_input[i], batchSize * INPUT_SIZE * sizeof(float)));
        CHECK_CUDA_ERROR(hipMalloc(&d_batch_hidden[i], batchSize * HIDDEN_SIZE * sizeof(float)));
        CHECK_CUDA_ERROR(hipMalloc(&d_batch_output[i], batchSize * OUTPUT_SIZE * sizeof(float)));
        CHECK_CUDA_ERROR(hipMalloc(&d_batch_target[i], batchSize * OUTPUT_SIZE * sizeof(float)));
    }
    
    // Allocate metrics memory
    float *d_loss;
    int *d_correct;
    CHECK_CUDA_ERROR(hipMalloc(&d_loss, sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_correct, sizeof(int)));
    
    // Use page-locked memory for faster transfers (also double-buffered)
    float *h_batch_data[2];
    CHECK_CUDA_ERROR(hipHostMalloc(&h_batch_data[0], batchSize * (INPUT_SIZE + OUTPUT_SIZE) * sizeof(float)));
    CHECK_CUDA_ERROR(hipHostMalloc(&h_batch_data[1], batchSize * (INPUT_SIZE + OUTPUT_SIZE) * sizeof(float)));
    
    float *h_batch_input[2], *h_batch_target[2];
    for (int i = 0; i < 2; i++) {
        h_batch_input[i] = h_batch_data[i];
        h_batch_target[i] = h_batch_data[i] + batchSize * INPUT_SIZE;
    }
    
    // Create index array for shuffling
    int* indices = (int*)malloc(numImages * sizeof(int));
    for (int i = 0; i < numImages; i++) {
        indices[i] = i;
    }
    
    // Create two CUDA streams for overlapping operations
    hipStream_t stream[2];
    hipStreamCreate(&stream[0]);
    hipStreamCreate(&stream[1]);
    
    for (int epoch = 0; epoch < EPOCHS; epoch++) {
        // Reset metrics
        float h_loss = 0.0f;
        int h_correct = 0;
        CHECK_CUDA_ERROR(hipMemcpy(d_loss, &h_loss, sizeof(float), hipMemcpyHostToDevice));
        CHECK_CUDA_ERROR(hipMemcpy(d_correct, &h_correct, sizeof(int), hipMemcpyHostToDevice));
        
        clock_t epoch_start = clock();
        float transferTime = 0, forwardTime = 0, backwardTime = 0;
        
        // Shuffle data indices
        for (int i = numImages - 1; i > 0; i--) {
            int j = rand() % (i + 1);
            int temp = indices[i];
            indices[i] = indices[j];
            indices[j] = temp;
        }
        
        // Prepare first batch before the main loop
        int current_batch_size = min(batchSize, numImages);
        int idx_buf = 0;  // Start with buffer 0
        
        // Prepare the first batch data
        for (int i = 0; i < current_batch_size; i++) {
            int idx = indices[i];
            
            memcpy(h_batch_input[idx_buf] + i * INPUT_SIZE, 
                   h_images[idx], 
                   INPUT_SIZE * sizeof(float));
                   
            memcpy(h_batch_target[idx_buf] + i * OUTPUT_SIZE, 
                   h_labels[idx], 
                   OUTPUT_SIZE * sizeof(float));
        }
        
        // Start first batch transfer
        hipEventRecord(start, stream[idx_buf]);
        hipMemcpyAsync(d_batch_input[idx_buf], h_batch_input[idx_buf],
            current_batch_size * INPUT_SIZE * sizeof(float), 
            hipMemcpyHostToDevice, stream[idx_buf]);
            
        hipMemcpyAsync(d_batch_target[idx_buf], h_batch_target[idx_buf],
            current_batch_size * OUTPUT_SIZE * sizeof(float), 
            hipMemcpyHostToDevice, stream[idx_buf]);
        hipEventRecord(stop, stream[idx_buf]);
        
        // Train in batches with prefetching
        for (int batch = 0; batch < numBatches; batch++) {
            int start_idx = batch * batchSize;
            current_batch_size = min(batchSize, numImages - start_idx);
            
            // Current buffer index
            int curr_buf = idx_buf;
            
            // Next buffer index (for prefetching)
            idx_buf = 1 - idx_buf;
            
            // Wait for current transfer to complete
            hipEventSynchronize(stop);
            hipEventElapsedTime(&milliseconds, start, stop);
            transferTime += milliseconds;
            
            // Start preparing next batch data (if not the last batch)
            if (batch + 1 < numBatches) {
                int next_start_idx = (batch + 1) * batchSize;
                int next_batch_size = min(batchSize, numImages - next_start_idx);
                
                // Prepare next batch in the other buffer
                for (int i = 0; i < next_batch_size; i++) {
                    int idx = indices[next_start_idx + i];
                    
                    memcpy(h_batch_input[idx_buf] + i * INPUT_SIZE, 
                           h_images[idx], 
                           INPUT_SIZE * sizeof(float));
                           
                    memcpy(h_batch_target[idx_buf] + i * OUTPUT_SIZE, 
                           h_labels[idx], 
                           OUTPUT_SIZE * sizeof(float));
                }
                
                // Start next batch transfer (overlapped with current computation)
                hipEventRecord(start, stream[idx_buf]);
                hipMemcpyAsync(d_batch_input[idx_buf], h_batch_input[idx_buf],
                    next_batch_size * INPUT_SIZE * sizeof(float), 
                    hipMemcpyHostToDevice, stream[idx_buf]);
                    
                hipMemcpyAsync(d_batch_target[idx_buf], h_batch_target[idx_buf],
                    next_batch_size * OUTPUT_SIZE * sizeof(float), 
                    hipMemcpyHostToDevice, stream[idx_buf]);
                hipEventRecord(stop, stream[idx_buf]);
            }
            
            // Process current batch
            hipEventRecord(start, stream[curr_buf]);
            forwardBatch(net, d_batch_input[curr_buf], d_batch_hidden[curr_buf], 
                        d_batch_output[curr_buf], current_batch_size, stream[curr_buf]);
                        
            calculateBatchLossAccuracy<<<current_batch_size, BLOCK_SIZE, 0, stream[curr_buf]>>>(
                d_batch_output[curr_buf], d_batch_target[curr_buf], d_loss, d_correct, current_batch_size);
            hipEventRecord(stop, stream[curr_buf]);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&milliseconds, start, stop);
            forwardTime += milliseconds;
            
            // Time backward pass
            hipEventRecord(start, stream[curr_buf]);
            batchBackward(net, d_batch_input[curr_buf], d_batch_hidden[curr_buf], 
                         d_batch_output[curr_buf], d_batch_target[curr_buf], current_batch_size);
            hipEventRecord(stop, stream[curr_buf]);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&milliseconds, start, stop);
            backwardTime += milliseconds;
        }
        
        // Get final metrics
        CHECK_CUDA_ERROR(hipMemcpy(&h_loss, d_loss, sizeof(float), hipMemcpyDeviceToHost));
        CHECK_CUDA_ERROR(hipMemcpy(&h_correct, d_correct, sizeof(int), hipMemcpyDeviceToHost));
        
        printf("Epoch %d - Loss: %.4f - Train Accuracy: %.2f%% - Time: %.3fs\n",
               epoch + 1, h_loss / numImages, (h_correct / (float)numImages) * 100,
               (float)(clock() - epoch_start) / CLOCKS_PER_SEC);
               
        printf("  Transfer: %.2f ms, Forward: %.2f ms, Backward: %.2f ms\n",
               transferTime, forwardTime, backwardTime);
    }
    
    // Clean up
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    // Clean up streams
    hipStreamDestroy(stream[0]);
    hipStreamDestroy(stream[1]);
    
    // Free double-buffered memory
    for (int i = 0; i < 2; i++) {
        hipFree(d_batch_input[i]);
        hipFree(d_batch_hidden[i]);
        hipFree(d_batch_output[i]);
        hipFree(d_batch_target[i]);
        hipHostFree(h_batch_data[i]);
    }
    
    hipFree(d_loss);
    hipFree(d_correct);
    free(indices);
}

// Free network memory
void freeNetwork(NeuralNetwork* net) {
    // Free host memory
    free(net->h_W1);
    free(net->h_W2);
    free(net->h_b1);
    free(net->h_b2);

    // Free device memory
    hipFree(net->d_W1);
    hipFree(net->d_W2);
    hipFree(net->d_b1);
    hipFree(net->d_b2);

    free(net);
}

// Updated evaluate function
void evaluate(NeuralNetwork* net, float** h_images, float** h_labels, int numImages) {
    const int batchSize = BATCH_SIZE;
    
    // Allocate device memory for batches
    float *d_batch_input, *d_batch_hidden, *d_batch_output, *d_batch_target;
    CHECK_CUDA_ERROR(hipMalloc(&d_batch_input, batchSize * INPUT_SIZE * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_batch_hidden, batchSize * HIDDEN_SIZE * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_batch_output, batchSize * OUTPUT_SIZE * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_batch_target, batchSize * OUTPUT_SIZE * sizeof(float)));
    
    // For loss and accuracy tracking
    float *d_loss;
    int *d_correct;
    CHECK_CUDA_ERROR(hipMalloc(&d_loss, sizeof(float)));  // Allocate dummy loss buffer
    CHECK_CUDA_ERROR(hipMalloc(&d_correct, sizeof(int)));
    
    // Initialize counters
    float h_loss = 0.0f;
    int h_correct = 0;
    CHECK_CUDA_ERROR(hipMemcpy(d_loss, &h_loss, sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_correct, &h_correct, sizeof(int), hipMemcpyHostToDevice));
    
    // Use pinned memory for faster transfers
    float *h_batch_input, *h_batch_target;
    CHECK_CUDA_ERROR(hipHostMalloc(&h_batch_input, batchSize * INPUT_SIZE * sizeof(float)));
    CHECK_CUDA_ERROR(hipHostMalloc(&h_batch_target, batchSize * OUTPUT_SIZE * sizeof(float)));
    
    // Process test data in batches
    for (int batch_start = 0; batch_start < numImages; batch_start += batchSize) {
        int current_batch_size = min(batchSize, numImages - batch_start);
        
        // Pack batch data 
        for (int i = 0; i < current_batch_size; i++) {
            memcpy(h_batch_input + i * INPUT_SIZE, 
                   h_images[batch_start + i],
                   INPUT_SIZE * sizeof(float));
            
            memcpy(h_batch_target + i * OUTPUT_SIZE,
                   h_labels[batch_start + i],
                   OUTPUT_SIZE * sizeof(float));
        }
        
        // Copy to device
        CHECK_CUDA_ERROR(hipMemcpy(d_batch_input, h_batch_input,
            current_batch_size * INPUT_SIZE * sizeof(float), hipMemcpyHostToDevice));
        CHECK_CUDA_ERROR(hipMemcpy(d_batch_target, h_batch_target,
            current_batch_size * OUTPUT_SIZE * sizeof(float), hipMemcpyHostToDevice));
        
        // Forward pass
        forwardBatch(net, d_batch_input, d_batch_hidden, d_batch_output, current_batch_size);
        
        // Use d_loss instead of nullptr - this was causing the error
        calculateBatchLossAccuracy<<<current_batch_size, BLOCK_SIZE>>>(
            d_batch_output, d_batch_target, d_loss, d_correct, current_batch_size);
    }
    
    // Get final accuracy
    CHECK_CUDA_ERROR(hipMemcpy(&h_correct, d_correct, sizeof(int), hipMemcpyDeviceToHost));
    
    // Clean up resources
    hipFree(d_batch_input);
    hipFree(d_batch_hidden);
    hipFree(d_batch_output);
    hipFree(d_batch_target);
    hipFree(d_loss);
    hipFree(d_correct);
    hipHostFree(h_batch_input);
    hipHostFree(h_batch_target);
    
    printf("Test Accuracy: %.2f%%\n", (h_correct / (float)numImages) * 100);
}
// Memory-mapped file loading for improved I/O performance
float** loadMNISTImagesOptimized(const char* filename, int numImages) {
    int fd = open(filename, O_RDONLY);
    if (fd < 0) {
        printf("Error opening %s\n", filename);
        exit(1);
    }
    
    // Get file size
    struct stat sb;
    if (fstat(fd, &sb) < 0) {
        printf("Error getting file size\n");
        close(fd);
        exit(1);
    }
    
    // Map file into memory
    unsigned char* file_data = (unsigned char*)mmap(
        NULL, sb.st_size, PROT_READ, MAP_PRIVATE, fd, 0);
    if (file_data == MAP_FAILED) {
        printf("Error mapping file\n");
        close(fd);
        exit(1);
    }
    
    // Allocate memory for images
    float** images = allocateMatrix(numImages, INPUT_SIZE);
    
    // Skip header (16 bytes)
    unsigned char* data_ptr = file_data + 16;
    
    // Process images with OpenMP
    #pragma omp parallel for
    for (int i = 0; i < numImages; i++) {
        for (int j = 0; j < INPUT_SIZE; j++) {
            images[i][j] = data_ptr[i * INPUT_SIZE + j] / 255.0f;
        }
    }
    
    // Unmap and close file
    munmap(file_data, sb.st_size);
    close(fd);
    
    return images;
}

// Memory-mapped label loading
float** loadMNISTLabelsOptimized(const char* filename, int numLabels) {
    int fd = open(filename, O_RDONLY);
    if (fd < 0) {
        printf("Error opening %s\n", filename);
        exit(1);
    }
    
    struct stat sb;
    fstat(fd, &sb);
    
    unsigned char* file_data = (unsigned char*)mmap(
        NULL, sb.st_size, PROT_READ, MAP_PRIVATE, fd, 0);
    if (file_data == MAP_FAILED) {
        printf("Error mapping file\n");
        close(fd);
        exit(1);
    }
    
    float** labels = allocateMatrix(numLabels, OUTPUT_SIZE);
    
    // Skip header (8 bytes)
    unsigned char* data_ptr = file_data + 8;
    
    #pragma omp parallel for
    for (int i = 0; i < numLabels; i++) {
        unsigned char label = data_ptr[i];
        for (int j = 0; j < OUTPUT_SIZE; j++) {
            labels[i][j] = (j == label) ? 1.0f : 0.0f;
        }
    }
    
    munmap(file_data, sb.st_size);
    close(fd);
    
    return labels;
}
// Main function remains similar but with float arrays instead of double
int main() {
    printf("MNIST Neural Network\n\n");
    clock_t total_start = clock();

    // Measure time for loading data
    clock_t start = clock();
    
    // Load data using memory-mapped I/O (faster than thread-based loading)
    float **train_images = loadMNISTImagesOptimized("../data/train-images.idx3-ubyte", 60000);
    float **train_labels = loadMNISTLabelsOptimized("../data/train-labels.idx1-ubyte", 60000);
    float **test_images = loadMNISTImagesOptimized("../data/t10k-images.idx3-ubyte", 10000);
    float **test_labels = loadMNISTLabelsOptimized("../data/t10k-labels.idx1-ubyte", 10000);
    
    clock_t end = clock();
    printf("Time to load data: %.3fs\n", (double)(end - start) / CLOCKS_PER_SEC);
    
    // Measure time for training
    start = clock();
    NeuralNetwork* net = createNetwork();
    train(net, train_images, train_labels, 60000);
    end = clock();
    printf("Time to train: %.3fs\n", (double)(end - start) / CLOCKS_PER_SEC);

    // Measure time for evaluation
    start = clock();
    evaluate(net, test_images, test_labels, 10000);
    end = clock();
    printf("Time to evaluate: %.3fs\n", (double)(end - start) / CLOCKS_PER_SEC);

    // End measuring total execution time
    clock_t total_end = clock();
    printf("Total execution time: %.3fs\n", (double)(total_end - total_start) / CLOCKS_PER_SEC);

    // Cleanup - CPU-based as it's more efficient than GPU for memory management
    freeNetwork(net);
    
    // Free training and test data
    for (int i = 0; i < 60000; i++) {
        free(train_images[i]);
        free(train_labels[i]);
    }
    for (int i = 0; i < 10000; i++) {
        free(test_images[i]);
        free(test_labels[i]);
    }
    free(train_images);
    free(train_labels);
    free(test_images);
    free(test_labels);
    
    return 0;
}
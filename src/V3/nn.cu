#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#define INPUT_SIZE 784
#define HIDDEN_SIZE 128
#define OUTPUT_SIZE 10
#define LEARNING_RATE 0.01f
#define EPOCHS 3
#define BATCH_SIZE 64
#define NUM_CLASSES 10
#define BLOCK_SIZE 256

// Error checking macro
#define CHECK_CUDA_ERROR(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA Error in %s:%d: %s\n", __FILE__, __LINE__, \
                    hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

// Neural network structure with mixed precision support
typedef struct {
    // Master weights (FP32)
    float *d_W1, *d_W2;    // Device weights
    float *d_b1, *d_b2;    // Device biases
    
    // Half precision copies for computation (FP16)
    half *d_W1_half, *d_W2_half;  // Half precision weights
    half *d_b1_half, *d_b2_half;  // Half precision biases
    
    // Host weights (only needed for initialization and final results)
    float *h_W1, *h_W2;    
    float *h_b1, *h_b2;    
} NeuralNetwork;


// Kernel to convert float array to half precision
__global__ void convertFloatToHalfArray(float* src, half* dst, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        dst[idx] = __float2half(src[idx]);
    }
}

// Kernel to convert half array to float precision
__global__ void convertHalfToFloatArray(half* src, float* dst, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        dst[idx] = __half2float(src[idx]);
    }
}

// Fused kernel for matrix multiplication + ReLU activation with mixed precision
__global__ void batchFCReluKernelMixedPrecision(half* weights, half* inputs, half* outputs, half* bias,
                                            int output_size, int input_size, int batch_size) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int batch = blockIdx.z;
    
    if (batch < batch_size && row < output_size) {
        // Use float for accumulation (higher precision)
        float sum = __half2float(bias[row]);
        
        // Calculate matrix multiplication with half precision inputs
        // but float accumulation for better numerical stability
        for (int i = 0; i < input_size; i++) {
            sum += __half2float(weights[row * input_size + i]) * 
                   __half2float(inputs[batch * input_size + i]);
        }
        
        // Apply ReLU and convert back to half precision
        outputs[batch * output_size + row] = __float2half(fmaxf(0.0f, sum));
    }
}

// Kernel for matrix multiplication + linear (no activation) with mixed precision
__global__ void batchFCKernelMixedPrecision(half* weights, half* inputs, half* outputs, half* bias,
                                        int output_size, int input_size, int batch_size) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int batch = blockIdx.z;
    
    if (batch < batch_size && row < output_size) {
        // Use float for accumulation
        float sum = __half2float(bias[row]);
        
        // Matrix multiplication with FP32 accumulation
        for (int i = 0; i < input_size; i++) {
            sum += __half2float(weights[row * input_size + i]) * 
                   __half2float(inputs[batch * input_size + i]);
        }
        
        // Store result in half precision
        outputs[batch * output_size + row] = __float2half(sum);
    }
}

__global__ void calculateBatchLossAccuracyMixedPrecision(half* d_batch_output, half* d_batch_target, 
                                                       float* d_loss, int* d_correct, int batchSize) {
    __shared__ float batch_loss[BLOCK_SIZE];
    __shared__ int batch_correct[BLOCK_SIZE];
    
    int tid = threadIdx.x;
    int batch_idx = blockIdx.x;
    
    batch_loss[tid] = 0.0f;
    batch_correct[tid] = 0;
    
    if (batch_idx < batchSize) {
        // Each thread handles one sample in the batch
        if (tid == 0) {
            half* output = d_batch_output + batch_idx * OUTPUT_SIZE;
            half* target = d_batch_target + batch_idx * OUTPUT_SIZE;
            
            // Find predicted class
            int pred = 0;
            for (int j = 1; j < OUTPUT_SIZE; j++) {
                if (__half2float(output[j]) > __half2float(output[pred])) {
                    pred = j;
                }
            }
            
            // Find actual class
            int actual = 0;
            for (int j = 1; j < OUTPUT_SIZE; j++) {
                if (__half2float(target[j]) > __half2float(target[actual])) {
                    actual = j;
                }
            }
            
            // Compute loss
            for (int j = 0; j < OUTPUT_SIZE; j++) {
                if (__half2float(target[j]) > 0.5f) {
                    batch_loss[tid] -= logf(fmaxf(__half2float(output[j]), 1e-7f));
                }
            }
            
            // Check if prediction was correct
            if (pred == actual) {
                batch_correct[tid] = 1;
            }
        }
    }
    
    __syncthreads();
    
    // Reduce within block
    for (int stride = blockDim.x/2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            batch_loss[tid] += batch_loss[tid + stride];
            batch_correct[tid] += batch_correct[tid + stride];
        }
        __syncthreads();
    }
    
    // Write results back
    if (tid == 0) {
        atomicAdd(d_loss, batch_loss[0]);
        atomicAdd(d_correct, batch_correct[0]);
    }
}
// Allocate memory for a matrix
float** allocateMatrix(int rows, int cols) {
    float** mat = (float**)malloc(rows * sizeof(float*));
    for (int i = 0; i < rows; i++) {
        mat[i] = (float*)malloc(cols * sizeof(float));
    }
    return mat;
}

NeuralNetwork* createNetwork() {
    NeuralNetwork* net = (NeuralNetwork*)malloc(sizeof(NeuralNetwork));
    
    // Allocate host memory (FP32 only)
    net->h_W1 = (float*)malloc(HIDDEN_SIZE * INPUT_SIZE * sizeof(float));
    net->h_W2 = (float*)malloc(OUTPUT_SIZE * HIDDEN_SIZE * sizeof(float));
    net->h_b1 = (float*)malloc(HIDDEN_SIZE * sizeof(float));
    net->h_b2 = (float*)malloc(OUTPUT_SIZE * sizeof(float));

    // Xavier/Glorot initialization
    float w1_scale = sqrtf(6.0f / (INPUT_SIZE + HIDDEN_SIZE));
    float w2_scale = sqrtf(6.0f / (HIDDEN_SIZE + OUTPUT_SIZE));

    for (int i = 0; i < HIDDEN_SIZE * INPUT_SIZE; i++)
        net->h_W1[i] = ((2.0f * (float)rand() / RAND_MAX) - 1.0f) * w1_scale;
    for (int i = 0; i < OUTPUT_SIZE * HIDDEN_SIZE; i++)
        net->h_W2[i] = ((2.0f * (float)rand() / RAND_MAX) - 1.0f) * w2_scale;
    for (int i = 0; i < HIDDEN_SIZE; i++)
        net->h_b1[i] = 0.0f;
    for (int i = 0; i < OUTPUT_SIZE; i++)
        net->h_b2[i] = 0.0f;

    // Allocate device memory for FP32 (master) weights
    CHECK_CUDA_ERROR(hipMalloc(&net->d_W1, HIDDEN_SIZE * INPUT_SIZE * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&net->d_W2, OUTPUT_SIZE * HIDDEN_SIZE * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&net->d_b1, HIDDEN_SIZE * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&net->d_b2, OUTPUT_SIZE * sizeof(float)));
    
    // Allocate device memory for FP16 (computation) weights
    CHECK_CUDA_ERROR(hipMalloc(&net->d_W1_half, HIDDEN_SIZE * INPUT_SIZE * sizeof(half)));
    CHECK_CUDA_ERROR(hipMalloc(&net->d_W2_half, OUTPUT_SIZE * HIDDEN_SIZE * sizeof(half)));
    CHECK_CUDA_ERROR(hipMalloc(&net->d_b1_half, HIDDEN_SIZE * sizeof(half)));
    CHECK_CUDA_ERROR(hipMalloc(&net->d_b2_half, OUTPUT_SIZE * sizeof(half)));

    // Copy FP32 data to device
    CHECK_CUDA_ERROR(hipMemcpy(net->d_W1, net->h_W1, 
        HIDDEN_SIZE * INPUT_SIZE * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(net->d_W2, net->h_W2,
        OUTPUT_SIZE * HIDDEN_SIZE * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(net->d_b1, net->h_b1,
        HIDDEN_SIZE * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(net->d_b2, net->h_b2,
        OUTPUT_SIZE * sizeof(float), hipMemcpyHostToDevice));
        
    // Convert and copy to half precision versions
    convertFloatToHalfArray<<<(HIDDEN_SIZE * INPUT_SIZE + 255) / 256, 256>>>(
        net->d_W1, net->d_W1_half, HIDDEN_SIZE * INPUT_SIZE);
    convertFloatToHalfArray<<<(OUTPUT_SIZE * HIDDEN_SIZE + 255) / 256, 256>>>(
        net->d_W2, net->d_W2_half, OUTPUT_SIZE * HIDDEN_SIZE);
    convertFloatToHalfArray<<<(HIDDEN_SIZE + 255) / 256, 256>>>(
        net->d_b1, net->d_b1_half, HIDDEN_SIZE);
    convertFloatToHalfArray<<<(OUTPUT_SIZE + 255) / 256, 256>>>(
        net->d_b2, net->d_b2_half, OUTPUT_SIZE);

    return net;
}
__global__ void batchComputeGradientsMixedPrecision(half* d_batch_output, half* d_batch_target, 
                                                   half* d_batch_hidden, half* d_batch_input,
                                                   float* d_W2_grad, float* d_b2_grad,
                                                   float* d_W1_grad, float* d_b1_grad,
                                                   half* d_W2_half, half* d_W1_half,
                                                   int batchSize) {
    int tid = threadIdx.x;
    int batch = blockIdx.x;
    
    if (batch >= batchSize) return;
    
    // Get pointers to this batch's data
    half* output = d_batch_output + batch * OUTPUT_SIZE;
    half* target = d_batch_target + batch * OUTPUT_SIZE;
    half* hidden = d_batch_hidden + batch * HIDDEN_SIZE;
    half* input = d_batch_input + batch * INPUT_SIZE;
    
    // Use shared memory to compute and store output errors (as float)
    __shared__ float output_errors[32];  // Enough for OUTPUT_SIZE=10
    
    // Calculate output errors in parallel, converting to float for better precision
    if (tid < OUTPUT_SIZE) {
        output_errors[tid] = __half2float(output[tid]) - __half2float(target[tid]);
        
        // Update bias gradient
        atomicAdd(&d_b2_grad[tid], output_errors[tid]);
    }
    __syncthreads();
    
    // Each thread updates a subset of W2 gradients
    for (int i = tid; i < OUTPUT_SIZE * HIDDEN_SIZE; i += blockDim.x) {
        int out_idx = i / HIDDEN_SIZE;
        int hid_idx = i % HIDDEN_SIZE;
        atomicAdd(&d_W2_grad[i], output_errors[out_idx] * __half2float(hidden[hid_idx]));
    }
    
    // Compute hidden layer errors
    __shared__ float hidden_errors[128];  // Maximum HIDDEN_SIZE we'd expect
    
    // First initialize all to zero
    for (int i = tid; i < HIDDEN_SIZE; i += blockDim.x) {
        hidden_errors[i] = 0.0f;
    }
    __syncthreads();
    
    // Compute hidden errors in parallel across threads
    for (int i = 0; i < OUTPUT_SIZE; i++) {
        for (int j = tid; j < HIDDEN_SIZE; j += blockDim.x) {
            hidden_errors[j] += output_errors[i] * __half2float(d_W2_half[i * HIDDEN_SIZE + j]);
        }
    }
    __syncthreads();
    
    // Apply ReLU derivative and update bias gradients
    for (int i = tid; i < HIDDEN_SIZE; i += blockDim.x) {
        // Apply ReLU derivative - output is positive if hidden value is positive
        hidden_errors[i] *= (__half2float(hidden[i]) > 0.0f);
        atomicAdd(&d_b1_grad[i], hidden_errors[i]);
    }
    __syncthreads();
    
    // Each thread updates a subset of W1 gradients
    for (int i = tid; i < HIDDEN_SIZE * INPUT_SIZE; i += blockDim.x) {
        int hid_idx = i / INPUT_SIZE;
        int in_idx = i % INPUT_SIZE;
        atomicAdd(&d_W1_grad[i], hidden_errors[hid_idx] * __half2float(input[in_idx]));
    }
}
// Optimized parameter update kernel
__global__ void batchUpdateParametersOptimized(float* d_W, float* d_b, float* d_W_grad, 
                                           float* d_b_grad, int rows, int cols, 
                                           float learning_rate, int batchSize) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Each thread updates multiple weights using grid-stride loop
    for (int i = tid; i < rows * cols; i += blockDim.x * gridDim.x) {
        d_W[i] -= (learning_rate / batchSize) * d_W_grad[i];
        d_W_grad[i] = 0.0f;  // Reset gradient for next batch
    }
    
    // Update biases with efficient access pattern
    if (tid < rows) {
        d_b[tid] -= (learning_rate / batchSize) * d_b_grad[tid];
        d_b_grad[tid] = 0.0f;  // Reset gradient for next batch
    }
}

// Backward pass with mixed precision
void batchBackwardMixedPrecision(NeuralNetwork* net, half* d_batch_input, half* d_batch_hidden, 
                             half* d_batch_output, half* d_batch_target, int batchSize) {
    // We'll still maintain gradient buffers in FP32 for better stability
    static float *d_W1_grad, *d_W2_grad, *d_b1_grad, *d_b2_grad;
    static bool gradients_initialized = false;
    
    // Initialize gradient buffers if first call
    if (!gradients_initialized) {
        CHECK_CUDA_ERROR(hipMalloc(&d_W1_grad, HIDDEN_SIZE * INPUT_SIZE * sizeof(float)));
        CHECK_CUDA_ERROR(hipMalloc(&d_W2_grad, OUTPUT_SIZE * HIDDEN_SIZE * sizeof(float)));
        CHECK_CUDA_ERROR(hipMalloc(&d_b1_grad, HIDDEN_SIZE * sizeof(float)));
        CHECK_CUDA_ERROR(hipMalloc(&d_b2_grad, OUTPUT_SIZE * sizeof(float)));
        
        gradients_initialized = true;
    }
    
    // Clear gradients
    hipMemset(d_W1_grad, 0, HIDDEN_SIZE * INPUT_SIZE * sizeof(float));
    hipMemset(d_W2_grad, 0, OUTPUT_SIZE * HIDDEN_SIZE * sizeof(float));
    hipMemset(d_b1_grad, 0, HIDDEN_SIZE * sizeof(float));
    hipMemset(d_b2_grad, 0, OUTPUT_SIZE * sizeof(float));
    
    // Compute gradients with mixed precision kernel
    dim3 blockDim(256);
    dim3 gridDim(batchSize);
    
    // This kernel will compute gradients using half precision inputs
    // but accumulate in float precision for the weight updates
    batchComputeGradientsMixedPrecision<<<gridDim, blockDim>>>(
        d_batch_output, d_batch_target, d_batch_hidden, d_batch_input,
        d_W2_grad, d_b2_grad, d_W1_grad, d_b1_grad,
        net->d_W2_half, net->d_W1_half, batchSize
    );
    
    // Update parameters using FP32 master weights
    batchUpdateParametersOptimized<<<32, 256>>>(
        net->d_W1, net->d_b1, d_W1_grad, d_b1_grad,
        HIDDEN_SIZE, INPUT_SIZE, LEARNING_RATE, batchSize
    );
    
    batchUpdateParametersOptimized<<<32, 256>>>(
        net->d_W2, net->d_b2, d_W2_grad, d_b2_grad,
        OUTPUT_SIZE, HIDDEN_SIZE, LEARNING_RATE, batchSize
    );
    
    // After updating the FP32 master weights, update the FP16 copies
    convertFloatToHalfArray<<<(HIDDEN_SIZE * INPUT_SIZE + 255) / 256, 256>>>(
        net->d_W1, net->d_W1_half, HIDDEN_SIZE * INPUT_SIZE);
    convertFloatToHalfArray<<<(OUTPUT_SIZE * HIDDEN_SIZE + 255) / 256, 256>>>(
        net->d_W2, net->d_W2_half, OUTPUT_SIZE * HIDDEN_SIZE);
    convertFloatToHalfArray<<<(HIDDEN_SIZE + 255) / 256, 256>>>(
        net->d_b1, net->d_b1_half, HIDDEN_SIZE);
    convertFloatToHalfArray<<<(OUTPUT_SIZE + 255) / 256, 256>>>(
        net->d_b2, net->d_b2_half, OUTPUT_SIZE);
}

// Optimized softmax for half precision
__global__ void batchSoftmaxSmallKernelMixedPrecision(half* x, int size, int batchSize) {
    int batch = blockIdx.x;
    int tid = threadIdx.x;
    
    if (batch < batchSize) {
        // Get pointer to this batch's data
        half* batch_data = x + batch * size;
        
        // Use shared memory for this small array
        __shared__ float data[32];  // Convert to float for computation
        __shared__ float max_val;
        __shared__ float sum_val;
        
        // Load and convert data to float for better precision
        if (tid < size) {
            data[tid] = __half2float(batch_data[tid]);
        }
        __syncthreads();
        
        // Find maximum with thread 0
        if (tid == 0) {
            max_val = data[0];
            for (int i = 1; i < size; i++) {
                max_val = fmaxf(max_val, data[i]);
            }
        }
        __syncthreads();
        
        // Compute exp(x - max) in float precision
        if (tid < size) {
            data[tid] = expf(data[tid] - max_val);
        }
        __syncthreads();
        
        // Compute sum with reduction
        if (tid == 0) {
            sum_val = 0.0f;
            for (int i = 0; i < size; i++) {
                sum_val += data[i];
            }
        }
        __syncthreads();
        
        // Normalize and write back to half precision
        if (tid < size) {
            batch_data[tid] = __float2half(data[tid] / sum_val);
        }
    }
}
// Forward pass using mixed precision
void forwardBatchMixedPrecision(NeuralNetwork* net, half* d_batch_input, half* d_batch_hidden, 
    half* d_batch_output, int batchSize, hipStream_t stream = 0) {
// Set block dimensions for FC layers
dim3 blockDim(1, 256);  // Using y dimension for row parallelism

// First layer: FC + ReLU fused (using half precision)
dim3 gridDim1(1, (HIDDEN_SIZE + blockDim.y - 1) / blockDim.y, batchSize);
batchFCReluKernelMixedPrecision<<<gridDim1, blockDim, 0, stream>>>(
net->d_W1_half, d_batch_input, d_batch_hidden, 
net->d_b1_half, HIDDEN_SIZE, INPUT_SIZE, batchSize);

// Second layer: FC (using half precision)
dim3 gridDim2(1, (OUTPUT_SIZE + blockDim.y - 1) / blockDim.y, batchSize);
batchFCKernelMixedPrecision<<<gridDim2, blockDim, 0, stream>>>(
net->d_W2_half, d_batch_hidden, d_batch_output,
net->d_b2_half, OUTPUT_SIZE, HIDDEN_SIZE, batchSize);

// Apply optimized softmax for small output vectors
// The softmax kernel will need to be updated to work with half precision
batchSoftmaxSmallKernelMixedPrecision<<<batchSize, 32, 0, stream>>>(
d_batch_output, OUTPUT_SIZE, batchSize);
}
// New train function with mixed precision
void trainMixedPrecision(NeuralNetwork* net, float** h_images, float** h_labels, int numImages) {
    const int batchSize = BATCH_SIZE;
    const int numBatches = (numImages + batchSize - 1) / batchSize;
    
    // CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    
    // Allocate device memory in half precision for activations
    half *d_batch_input_half, *d_batch_hidden_half, *d_batch_output_half, *d_batch_target_half;
    CHECK_CUDA_ERROR(hipMalloc(&d_batch_input_half, batchSize * INPUT_SIZE * sizeof(half)));
    CHECK_CUDA_ERROR(hipMalloc(&d_batch_hidden_half, batchSize * HIDDEN_SIZE * sizeof(half)));
    CHECK_CUDA_ERROR(hipMalloc(&d_batch_output_half, batchSize * OUTPUT_SIZE * sizeof(half)));
    CHECK_CUDA_ERROR(hipMalloc(&d_batch_target_half, batchSize * OUTPUT_SIZE * sizeof(half)));
    
    // We need float buffers for initial data loading
    float *d_batch_input_float, *d_batch_target_float;
    CHECK_CUDA_ERROR(hipMalloc(&d_batch_input_float, batchSize * INPUT_SIZE * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_batch_target_float, batchSize * OUTPUT_SIZE * sizeof(float)));
    
    // Metrics are still in float precision
    float *d_loss;
    int *d_correct;
    CHECK_CUDA_ERROR(hipMalloc(&d_loss, sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_correct, sizeof(int)));
    
    // Use page-locked memory for faster transfers
    float *h_batch_data;  // Combined buffer for input and target
    size_t input_bytes = batchSize * INPUT_SIZE * sizeof(float);
    size_t target_bytes = batchSize * OUTPUT_SIZE * sizeof(float);
    CHECK_CUDA_ERROR(hipHostMalloc(&h_batch_data, input_bytes + target_bytes));
    
    float *h_batch_input = h_batch_data;
    float *h_batch_target = h_batch_data + batchSize * INPUT_SIZE;
    
    // Create index array for shuffling
    int* indices = (int*)malloc(numImages * sizeof(int));
    for (int i = 0; i < numImages; i++) {
        indices[i] = i;
    }
    
    // Create CUDA stream
    hipStream_t stream;
    hipStreamCreate(&stream);
    
    for (int epoch = 0; epoch < EPOCHS; epoch++) {
        // Reset metrics
        float h_loss = 0.0f;
        int h_correct = 0;
        CHECK_CUDA_ERROR(hipMemcpy(d_loss, &h_loss, sizeof(float), hipMemcpyHostToDevice));
        CHECK_CUDA_ERROR(hipMemcpy(d_correct, &h_correct, sizeof(int), hipMemcpyHostToDevice));
        
        clock_t epoch_start = clock();
        float transferTime = 0, forwardTime = 0, backwardTime = 0;
        
        // Shuffle data indices
        for (int i = numImages - 1; i > 0; i--) {
            int j = rand() % (i + 1);
            int temp = indices[i];
            indices[i] = indices[j];
            indices[j] = temp;
        }
        
        // Train in batches
        for (int batch = 0; batch < numBatches; batch++) {
            int start_idx = batch * batchSize;
            int current_batch_size = min(batchSize, numImages - start_idx);
            
            // Prepare batch data in coalesced order
            for (int i = 0; i < current_batch_size; i++) {
                int idx = indices[start_idx + i];
                
                memcpy(h_batch_input + i * INPUT_SIZE, 
                       h_images[idx], 
                       INPUT_SIZE * sizeof(float));
                       
                memcpy(h_batch_target + i * OUTPUT_SIZE, 
                       h_labels[idx], 
                       OUTPUT_SIZE * sizeof(float));
            }
            
            // Time transfer
            hipEventRecord(start, stream);
            
            // Transfer data to FP32 buffers first
            hipMemcpyAsync(d_batch_input_float, h_batch_input,
                current_batch_size * INPUT_SIZE * sizeof(float), 
                hipMemcpyHostToDevice, stream);
                
            hipMemcpyAsync(d_batch_target_float, h_batch_target,
                current_batch_size * OUTPUT_SIZE * sizeof(float), 
                hipMemcpyHostToDevice, stream);
                
            // Convert to half precision
            convertFloatToHalfArray<<<(current_batch_size * INPUT_SIZE + 255) / 256, 256, 0, stream>>>(
                d_batch_input_float, d_batch_input_half, current_batch_size * INPUT_SIZE);
                
            convertFloatToHalfArray<<<(current_batch_size * OUTPUT_SIZE + 255) / 256, 256, 0, stream>>>(
                d_batch_target_float, d_batch_target_half, current_batch_size * OUTPUT_SIZE);
            
            hipEventRecord(stop, stream);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&milliseconds, start, stop);
            transferTime += milliseconds;
            
            // Time forward pass
            hipEventRecord(start, stream);
            
            forwardBatchMixedPrecision(net, d_batch_input_half, d_batch_hidden_half, 
                                    d_batch_output_half, current_batch_size, stream);
                                    
            calculateBatchLossAccuracyMixedPrecision<<<current_batch_size, BLOCK_SIZE, 0, stream>>>(
                d_batch_output_half, d_batch_target_half, d_loss, d_correct, current_batch_size);
                
            hipEventRecord(stop, stream);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&milliseconds, start, stop);
            forwardTime += milliseconds;
            
            // Time backward pass
            hipEventRecord(start, stream);
            
            batchBackwardMixedPrecision(net, d_batch_input_half, d_batch_hidden_half, 
                                    d_batch_output_half, d_batch_target_half, current_batch_size);
                                    
            hipEventRecord(stop, stream);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&milliseconds, start, stop);
            backwardTime += milliseconds;
        }
        
        // Get final metrics
        CHECK_CUDA_ERROR(hipMemcpy(&h_loss, d_loss, sizeof(float), hipMemcpyDeviceToHost));
        CHECK_CUDA_ERROR(hipMemcpy(&h_correct, d_correct, sizeof(int), hipMemcpyDeviceToHost));
        
        printf("Epoch %d - Loss: %.4f - Train Accuracy: %.2f%% - Time: %.3fs\n",
               epoch + 1, h_loss / numImages, (h_correct / (float)numImages) * 100,
               (float)(clock() - epoch_start) / CLOCKS_PER_SEC);
               
        printf("  Transfer: %.2f ms, Forward: %.2f ms, Backward: %.2f ms\n",
               transferTime, forwardTime, backwardTime);
    }
    
    // Clean up
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipStreamDestroy(stream);
    
    // Free half precision buffers
    hipFree(d_batch_input_half);
    hipFree(d_batch_hidden_half);
    hipFree(d_batch_output_half);
    hipFree(d_batch_target_half);
    
    // Free float buffers
    hipFree(d_batch_input_float);
    hipFree(d_batch_target_float);
    
    hipFree(d_loss);
    hipFree(d_correct);
    hipHostFree(h_batch_data);
    free(indices);
}

// Free network memory
void freeNetwork(NeuralNetwork* net) {
    // Free host memory
    free(net->h_W1);
    free(net->h_W2);
    free(net->h_b1);
    free(net->h_b2);

    // Free device memory
    hipFree(net->d_W1);
    hipFree(net->d_W2);
    hipFree(net->d_b1);
    hipFree(net->d_b2);

    free(net);
}

// Mixed precision evaluation function
void evaluateMixedPrecision(NeuralNetwork* net, float** h_images, float** h_labels, int numImages) {
    const int batchSize = BATCH_SIZE;
    
    // Allocate device memory for batches - both float and half precision
    float *d_batch_input_float, *d_batch_target_float;
    half *d_batch_input_half, *d_batch_hidden_half, *d_batch_output_half, *d_batch_target_half;
    
    // Allocate float precision buffers for data loading
    CHECK_CUDA_ERROR(hipMalloc(&d_batch_input_float, batchSize * INPUT_SIZE * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_batch_target_float, batchSize * OUTPUT_SIZE * sizeof(float)));
    
    // Allocate half precision buffers for computation
    CHECK_CUDA_ERROR(hipMalloc(&d_batch_input_half, batchSize * INPUT_SIZE * sizeof(half)));
    CHECK_CUDA_ERROR(hipMalloc(&d_batch_hidden_half, batchSize * HIDDEN_SIZE * sizeof(half)));
    CHECK_CUDA_ERROR(hipMalloc(&d_batch_output_half, batchSize * OUTPUT_SIZE * sizeof(half)));
    CHECK_CUDA_ERROR(hipMalloc(&d_batch_target_half, batchSize * OUTPUT_SIZE * sizeof(half)));
    
    // For metrics tracking
    float *d_loss;
    int *d_correct;
    CHECK_CUDA_ERROR(hipMalloc(&d_loss, sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_correct, sizeof(int)));
    
    // Initialize counters
    float h_loss = 0.0f;
    int h_correct = 0;
    CHECK_CUDA_ERROR(hipMemcpy(d_loss, &h_loss, sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_correct, &h_correct, sizeof(int), hipMemcpyHostToDevice));
    
    // Use pinned memory for faster transfers
    float *h_batch_data;
    size_t total_bytes = batchSize * (INPUT_SIZE + OUTPUT_SIZE) * sizeof(float);
    CHECK_CUDA_ERROR(hipHostMalloc(&h_batch_data, total_bytes));
    
    float *h_batch_input = h_batch_data;
    float *h_batch_target = h_batch_data + batchSize * INPUT_SIZE;
    
    // Create CUDA stream
    hipStream_t stream;
    hipStreamCreate(&stream);
    
    // Process test data in batches
    for (int batch_start = 0; batch_start < numImages; batch_start += batchSize) {
        int current_batch_size = min(batchSize, numImages - batch_start);
        
        // Pack batch data in coalesced order
        for (int i = 0; i < current_batch_size; i++) {
            memcpy(h_batch_input + i * INPUT_SIZE, 
                   h_images[batch_start + i],
                   INPUT_SIZE * sizeof(float));
            
            memcpy(h_batch_target + i * OUTPUT_SIZE,
                   h_labels[batch_start + i],
                   OUTPUT_SIZE * sizeof(float));
        }
        
        // Copy to device (float precision first)
        hipMemcpyAsync(d_batch_input_float, h_batch_input,
            current_batch_size * INPUT_SIZE * sizeof(float), 
            hipMemcpyHostToDevice, stream);
            
        hipMemcpyAsync(d_batch_target_float, h_batch_target,
            current_batch_size * OUTPUT_SIZE * sizeof(float), 
            hipMemcpyHostToDevice, stream);
        
        // Convert to half precision
        convertFloatToHalfArray<<<(current_batch_size * INPUT_SIZE + 255) / 256, 256, 0, stream>>>(
            d_batch_input_float, d_batch_input_half, current_batch_size * INPUT_SIZE);
            
        convertFloatToHalfArray<<<(current_batch_size * OUTPUT_SIZE + 255) / 256, 256, 0, stream>>>(
            d_batch_target_float, d_batch_target_half, current_batch_size * OUTPUT_SIZE);
        
        // Forward pass with half precision
        forwardBatchMixedPrecision(net, d_batch_input_half, d_batch_hidden_half, 
                                d_batch_output_half, current_batch_size, stream);
        
        // Calculate accuracy with half precision
        calculateBatchLossAccuracyMixedPrecision<<<current_batch_size, BLOCK_SIZE, 0, stream>>>(
            d_batch_output_half, d_batch_target_half, d_loss, d_correct, current_batch_size);
    }
    
    hipStreamSynchronize(stream);
    
    // Get final metrics
    CHECK_CUDA_ERROR(hipMemcpy(&h_correct, d_correct, sizeof(int), hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipMemcpy(&h_loss, d_loss, sizeof(float), hipMemcpyDeviceToHost));
    
    // Clean up resources
    hipFree(d_batch_input_float);
    hipFree(d_batch_target_float);
    hipFree(d_batch_input_half);
    hipFree(d_batch_hidden_half);
    hipFree(d_batch_output_half);
    hipFree(d_batch_target_half);
    hipFree(d_loss);
    hipFree(d_correct);
    hipHostFree(h_batch_data);
    hipStreamDestroy(stream);
    
    printf("Test Accuracy: %.2f%% - Loss: %.4f\n", 
           (h_correct / (float)numImages) * 100, h_loss / numImages);
}

// Read MNIST dataset
float** loadMNISTImages(const char* filename, int numImages) {
    FILE* file = fopen(filename, "rb");
    if (!file) {
        printf("Error opening %s\n", filename);
        exit(1);
    }
    fseek(file, 16, SEEK_SET);
    float** images = allocateMatrix(numImages, INPUT_SIZE);
    for (int i = 0; i < numImages; i++) {
        for (int j = 0; j < INPUT_SIZE; j++) {
            unsigned char pixel;

            // fread(&pixel, sizeof(unsigned char), 1, file);
            if (fread(&pixel, sizeof(unsigned char), 1, file) != 1) {
                fprintf(stderr, "Error: Failed to read pixel\n");
                fclose(file);
                exit(EXIT_FAILURE);
            }

            images[i][j] = pixel / 255.0;
        }
    }
    fclose(file);
    return images;
}


float** loadMNISTLabels(const char* filename, int numLabels) {
    FILE* file = fopen(filename, "rb");
    if (!file) {
        printf("Error opening %s\n", filename);
        exit(1);
    }
    fseek(file, 8, SEEK_SET);
    float** labels = allocateMatrix(numLabels, OUTPUT_SIZE);
    for (int i = 0; i < numLabels; i++) {
        unsigned char label;
        // fread(&label, sizeof(unsigned char), 1, file);
        if (fread(&label, sizeof(unsigned char), 1, file) != 1) {
            fprintf(stderr, "Error: Failed to read label\n");
            fclose(file);
            exit(EXIT_FAILURE);
        }

        for (int j = 0; j < OUTPUT_SIZE; j++) {
            labels[i][j] = (j == label) ? 1.0 : 0.0;
        }
    }
    fclose(file);
    return labels;
}


// Main function remains similar but with float arrays instead of double
int main() {
    printf("MNIST Neural Network\n\n");
    // Start measuring total execution time
    clock_t total_start = clock();

    // Measure time for loading data
    clock_t start = clock();
    float** train_images = loadMNISTImages("../data/train-images.idx3-ubyte", 60000);
    float** train_labels = loadMNISTLabels("../data/train-labels.idx1-ubyte", 60000);
    float** test_images = loadMNISTImages("../data/t10k-images.idx3-ubyte", 10000);
    float** test_labels = loadMNISTLabels("../data/t10k-labels.idx1-ubyte", 10000);
    clock_t end = clock();
    printf("Time to load data: %.3fs\n", (double)(end - start) / CLOCKS_PER_SEC);

    // Measure time for training
    start = clock();
    NeuralNetwork* net = createNetwork();
    trainMixedPrecision(net, train_images, train_labels, 60000);
    end = clock();
    printf("Time to train: %.3fs\n", (double)(end - start) / CLOCKS_PER_SEC);

    // Measure time for evaluation
    start = clock();
    evaluateMixedPrecision(net, test_images, test_labels, 10000);

    end = clock();
    printf("Time to evaluate: %.3fs\n", (double)(end - start) / CLOCKS_PER_SEC);

    // End measuring total execution time
    clock_t total_end = clock();
    printf("Total execution time: %.3fs\n", (double)(total_end - total_start) / CLOCKS_PER_SEC);

    freeNetwork(net);
    return 0;
}
